#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2022 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: BSD-3-Clause                                     */
/*----------------------------------------------------------------------------*/

#include "../offload/offload_runtime.h"

#if defined(__OFFLOAD) && !defined(__NO_OFFLOAD_DBM)

#include "../offload/offload_library.h"
#include "dbm_mempool.h"
#include "dbm_multiply_gpu.h"

#include <assert.h>
#include <stdio.h>

/*******************************************************************************
 * \brief Atomic add for doubles that also works prior to compute capability 6.
 * \author Ole Schuett
 ******************************************************************************/
__device__ static void atomicAddDouble(double *address, double val) {
  if (val == 0.0)
    return;

#if __CUDA_ARCH__ >= 600
  atomicAdd(address, val); // part of gpu library
#else
  // https://docs.nvidia.com/gpu/gpu-c-programming-guide/index.html#atomic-functions
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));

    // Uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

#endif
}

/*******************************************************************************
 * \brief Internal routine for intializing the gpu backend.
 * \author Ole Schuett
 ******************************************************************************/
void dbm_multiply_gpu_start(const int max_batch_size, const int nshards,
                            dbm_shard_t *shards_c_host,
                            dbm_multiply_gpu_context_t *ctx) {
  // Select GPU device.
  offload_activate_chosen_device();

  ctx->nshards = nshards;
  ctx->shards_c_host = shards_c_host;
  ctx->max_batch_size = max_batch_size;
  offloadStreamCreate(&ctx->main_stream);

  // Allocate device storage for batches.
  const size_t size = nshards * max_batch_size * sizeof(dbm_task_t);
  ctx->batches_dev = (dbm_task_t *)dbm_mempool_device_malloc(size);

  // Allocate and upload shards of result matrix C.
  ctx->shards_c_dev =
      (dbm_shard_gpu_t *)malloc(nshards * sizeof(dbm_shard_gpu_t));
  for (int i = 0; i < nshards; i++) {
    offloadStreamCreate(&ctx->shards_c_dev[i].stream);
    ctx->shards_c_dev[i].data_size = ctx->shards_c_host[i].data_size;
    const size_t size = ctx->shards_c_dev[i].data_size * sizeof(double);
    ctx->shards_c_dev[i].data = (double *)dbm_mempool_device_malloc(size);
    offloadMemcpyAsyncHtoD(ctx->shards_c_dev[i].data,
                           ctx->shards_c_host[i].data, size,
                           ctx->shards_c_dev[i].stream);
  }
}

/*******************************************************************************
 * \brief Private routine for uploading a single pack onto the device.
 * \author Ole Schuett
 ******************************************************************************/
static void upload_pack(const dbm_pack_t *pack_host, dbm_pack_t *pack_dev,
                        const offloadStream_t stream) {

  const size_t size = pack_host->data_size * sizeof(double);
  if (pack_dev->data_size < pack_host->data_size) {
    dbm_mempool_free(pack_dev->data);
    pack_dev->data = (double *)dbm_mempool_device_malloc(size);
  }
  offloadMemcpyAsyncHtoD(pack_dev->data, pack_host->data, size, stream);
}

/*******************************************************************************
 * \brief Internal routine for uploading newly arrived packs onto the device.
 * \author Ole Schuett
 ******************************************************************************/
void dbm_multiply_gpu_upload_packs(const dbm_pack_t *pack_a,
                                   const dbm_pack_t *pack_b,
                                   dbm_multiply_gpu_context_t *ctx) {
  // Select GPU device.
  offload_activate_chosen_device();

  // Wait for all c-streams to complete before overwriting old packs.
  offloadEvent_t event;
  offloadEventCreate(&event);
  for (int i = 0; i < ctx->nshards; i++) {
    offloadEventRecord(event, ctx->shards_c_dev[i].stream);
    offloadStreamWaitEvent(ctx->main_stream, event, 0);
  }

  upload_pack(pack_a, &ctx->pack_a_dev, ctx->main_stream);
  upload_pack(pack_b, &ctx->pack_b_dev, ctx->main_stream);

  // Have all c-streams wait until new packs are uploaded.
  offloadEventRecord(event, ctx->main_stream);
  for (int i = 0; i < ctx->nshards; i++) {
    offloadStreamWaitEvent(ctx->shards_c_dev[i].stream, event, 0);
  }
  offloadEventDestroy(event);
}

/*******************************************************************************
 * \brief A very naive - but generic - matrix multiplication kernel.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void
process_batch_kernel(const bool transa, const bool transb, const double alpha,
                     const dbm_task_t *batch, const double *pack_a_data,
                     const double *pack_b_data, double *shard_c_data) {

  const dbm_task_t task = batch[blockIdx.x];
  const int lda = (transa) ? task.k : task.m;
  const int ldb = (transb) ? task.n : task.k;
  const int ldc = task.m;
  const double *data_a = &pack_a_data[task.offset_a];
  const double *data_b = &pack_b_data[task.offset_b];
  double *data_c = &shard_c_data[task.offset_c];

  for (int i = threadIdx.z; i < task.m; i += blockDim.z) {
    for (int j = threadIdx.y; j < task.n; j += blockDim.y) {
      for (int l = threadIdx.x; l < task.k; l += blockDim.x) {
        const int idx_a = (transa) ? i * lda + l : l * lda + i;
        const int idx_b = (transb) ? l * ldb + j : j * ldb + l;
        const int idx_c = j * ldc + i;
        const double val = alpha * data_a[idx_a] * data_b[idx_b];
        atomicAddDouble(&data_c[idx_c], val);
      }
    }
  }
}

/*******************************************************************************
 * \brief Internal routine for executing the tasks in given batch on the GPU.
 * \author Ole Schuett
 ******************************************************************************/
void dbm_multiply_gpu_process_batch(const int ntasks, const dbm_task_t *batch,
                                    const bool transa, const bool transb,
                                    const double alpha, const int kshard,
                                    dbm_multiply_gpu_context_t *ctx) {
  if (ntasks == 0) {
    return; // Nothing to do.
  }

  // Select GPU device.
  offload_activate_chosen_device();

  const dbm_shard_t *shard_c_host = &ctx->shards_c_host[kshard];
  dbm_shard_gpu_t *shard_c_dev = &ctx->shards_c_dev[kshard];

  // Upload new batch.
  dbm_task_t *batch_dev = &ctx->batches_dev[kshard * ctx->max_batch_size];
  const size_t size = ntasks * sizeof(dbm_task_t);
  offloadMemcpyAsyncHtoD(batch_dev, batch, size, shard_c_dev->stream);
  offloadEvent_t batch_uploaded;
  offloadEventCreate(&batch_uploaded);
  offloadEventRecord(batch_uploaded, shard_c_dev->stream);

  // Grow shard_c_dev->data if nessecary.
  if (shard_c_dev->data_size < shard_c_host->data_promised) {
    // TODO experiment with over-allocation.
    double *old_data_dev = shard_c_dev->data;
    const size_t old_size = shard_c_dev->data_size * sizeof(double);
    shard_c_dev->data_size = shard_c_host->data_promised;
    const size_t new_size = shard_c_dev->data_size * sizeof(double);
    shard_c_dev->data = (double *)dbm_mempool_device_malloc(new_size);
    offloadMemsetAsync(shard_c_dev->data, 0, new_size,
                       shard_c_dev->stream); // TODO: zero only tail
    offloadMemcpyAsyncDtoD(shard_c_dev->data, old_data_dev, old_size,
                           shard_c_dev->stream);
    // Wait for copy to complete before freeing old buffer.
    offloadStreamSynchronize(shard_c_dev->stream);
    dbm_mempool_free(old_data_dev);
  }

  // Launch kernel.
  const int nblocks = ntasks; // TODO tune launch parameters.
  const dim3 threads_per_block(4, 4, 4);
  const size_t smem_per_block = 0;
  process_batch_kernel<<<nblocks, threads_per_block, smem_per_block,
                         shard_c_dev->stream>>>(
      transa, transb, alpha, batch_dev, ctx->pack_a_dev.data,
      ctx->pack_b_dev.data, shard_c_dev->data);
  OFFLOAD_CHECK(offloadGetLastError());

  // Wait for batch to be uploaded before refilling it.
  offloadEventSynchronize(batch_uploaded);
  offloadEventDestroy(batch_uploaded);
}

/*******************************************************************************
 * \brief Internal routine for downloading results from the device.
 * \author Ole Schuett
 ******************************************************************************/
void dbm_multiply_gpu_download_results(dbm_multiply_gpu_context_t *ctx) {
  // Select GPU device.
  offload_activate_chosen_device();

#pragma omp parallel for schedule(dynamic)
  for (int i = 0; i < ctx->nshards; i++) {
    // Grow host buffer if nessecary.
    dbm_shard_t *shard_c_host = &ctx->shards_c_host[i];
    dbm_shard_allocate_promised_blocks(shard_c_host);

    // Download results from device.
    dbm_shard_gpu_t *shard_c_dev = &ctx->shards_c_dev[i];
    assert(shard_c_host->data_size == shard_c_dev->data_size);
    const size_t size = shard_c_dev->data_size * sizeof(double);
    offloadMemcpyAsyncDtoH(shard_c_host->data, shard_c_dev->data, size,
                           shard_c_dev->stream);
  }
}

/*******************************************************************************
 * \brief Internal routine for shutting down the gpu backend.
 * \author Ole Schuett
 ******************************************************************************/
void dbm_multiply_gpu_stop(dbm_multiply_gpu_context_t *ctx) {
  // Select GPU device.
  offload_activate_chosen_device();

  // Wait for completion, then free gpu ressources.
#pragma omp parallel for schedule(dynamic)
  for (int i = 0; i < ctx->nshards; i++) {
    dbm_shard_gpu_t *shard_c_dev = &ctx->shards_c_dev[i];
    offloadStreamSynchronize(shard_c_dev->stream);
    offloadStreamDestroy(shard_c_dev->stream);
    dbm_mempool_free(shard_c_dev->data);
  }
  free(ctx->shards_c_dev);

  dbm_mempool_free(ctx->pack_a_dev.data);
  dbm_mempool_free(ctx->pack_b_dev.data);
  dbm_mempool_free(ctx->batches_dev);
  offloadStreamDestroy(ctx->main_stream);
}

#endif // defined(__OFFLOAD) && !defined(__NO_OFFLOAD_DBM)

// EOF

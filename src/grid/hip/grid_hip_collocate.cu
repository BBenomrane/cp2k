
#include <hip/hip_runtime.h>
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2023 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: BSD-3-Clause                                     */
/*----------------------------------------------------------------------------*/

/*
 * Authors :
 - Dr Mathieu Taillefumier (ETH Zurich / CSCS)
 - Advanced Micro Devices, Inc.
*/

#if defined(__OFFLOAD_HIP) && !defined(__NO_OFFLOAD_GRID)

#include <algorithm>
#include <assert.h>
#include <climits>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

#include "grid_hip_internal_header.h"
#include "grid_hip_prepare_pab.h"

#if defined(_OMP_H)
#error "OpenMP should not be used in .cu files to accommodate HIP."
#endif

namespace rocm_backend {
/*******************************************************************************
 * \brief Decontracts the subblock, going from spherical to cartesian harmonics.
 ******************************************************************************/
template <typename T, bool IS_FUNC_AB>
__device__ __inline__ void block_to_cab(const kernel_params &params,
                                        const smem_task<T> &task, T *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The cartesian index runs over exponents and then over angular momentum.

  // This is a T matrix product. Since the pab block can be quite large the
  // two products are fused to conserve shared memory.
  const int tid =
      threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
  for (int i = 0; i < task.nsgf_setb; i++) {
    for (int j = 0; j < task.nsgf_seta; j++) {
      T block_val;
      if (task.block_transposed) {
        block_val = task.pab_block[j * task.nsgfb + i] * task.off_diag_twice;
      } else {
        block_val = task.pab_block[i * task.nsgfa + j] * task.off_diag_twice;
      }

      // fast path for common case
      // const int jco_start = task.first_cosetb;
      for (int jco = task.first_cosetb + tid / 8; jco < task.ncosetb;
           jco += 8) {
        const T sphib = task.sphib[i * task.maxcob + jco];
        for (int ico = task.first_coseta + (tid % 8); ico < task.ncoseta;
             ico += 8) {
          const T sphia = task.sphia[j * task.maxcoa + ico];
          const T pab_val = block_val * sphia * sphib;
          if (IS_FUNC_AB) {
            cab[jco * task.ncoseta + ico] += pab_val;
          } else {
            const auto a = coset_inv[ico];
            const auto b = coset_inv[jco];
            prepare_pab(params.func, a, b, task.zeta, task.zetb, pab_val,
                        task.n1, cab);
          }
        }
      }
      __syncthreads();
    }
  }
}

template <typename T, bool IS_FUNC_AB>
__global__ void calculate_coefficients(const kernel_params dev_) {
  __shared__ smem_task<T> task;
  if (dev_.tasks[dev_.first_task + blockIdx.x].skip_task)
    return;

  const int tid =
      threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);

  fill_smem_task_coef(dev_, dev_.first_task + blockIdx.x, task);
  extern __shared__ T shared_memory[];
  T *smem_cab = &shared_memory[dev_.smem_cab_offset];
  T *smem_alpha = &shared_memory[dev_.smem_alpha_offset];
  T *coef_ =
      &dev_.ptr_dev[2][dev_.tasks[dev_.first_task + blockIdx.x].coef_offset];

  for (int z = tid; z < task.n1 * task.n2;
       z += blockDim.x * blockDim.y * blockDim.z)
    smem_cab[z] = 0.0;

  __syncthreads();
  block_to_cab<T, IS_FUNC_AB>(dev_, task, smem_cab);
  __syncthreads();
  compute_alpha(task, smem_alpha);
  __syncthreads();
  cab_to_cxyz(task, smem_alpha, smem_cab, coef_);
}

/*
  \brief compute the real space representation of an operator expressed in the
  gaussian basis

  this kernel does the following operation

  n_{ijk} = \f[
  \sum_{p,\alpha,\beta,\gamma} C^p_{\alpha\beta\gamma} X_{\alpha,i} Y_{\beta, j}
  Z_{\gamma, k} \exp\left(- \eta (r_{ijk} - r_c) ^ 2\right)
  ]

  where $X_{\alpha,i}, Y_{\beta, j}, Z_{\gamma, k}$ are polynomials of degree
  $\alpha,\beta,\gamma$ and $r_{ijk}% a point (in cartesian coordinates) on a 3D
  grid. C^p_{\alpha\beta\gamma} are also constrained such that 0 <= \alpha +
  \beta + \gamma <= lmax. It means in practice that we need store (lmax + 1) *
  (lamax + 2) * (lmax + 3) / 6 coefficients all the other coefficients are zero

  to reduce computation, a spherical cutoff is applied such that all points
  $|r_{ijk} - r_c| > radius$ are not computed. The sum over p extends over all
  relevant pairs of gaussians (which are called task in the code).

  the kernel computes the polynomials and the gaussian then sums the result
  back to the grid.

  the coefficients $C^p_{\alpha\beta\gamma}$ are computed by
  calculate_coefficients. We only keep the non zero elements to same memory.
*/

template <typename T, typename T3, bool distributed__, bool orthorhombic_>
__global__
__launch_bounds__(64) void collocate_kernel(const kernel_params dev_) {
  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task_reduced<T, T3> task;

  if (dev_.tasks[dev_.first_task + blockIdx.x].skip_task)
    return;

  fill_smem_task_reduced(dev_, dev_.first_task + blockIdx.x, task);

  const int tid =
      threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);

  //  Alloc shared memory.
  extern __shared__ T coefs_[];

  T *coef_ =
      &dev_.ptr_dev[2][dev_.tasks[dev_.first_task + blockIdx.x].coef_offset];
  __shared__ T dh_[9], dh_inv_[9];

  // matrix from lattice coordinates to cartesian coordinates
  for (int i = tid; i < 9; i += blockDim.x * blockDim.y * blockDim.z) {
    dh_[i] = dev_.dh_[i];
  }

  // matrix from  cartesian coordinates to lattice coordinates.
  for (int i = tid; i < 9; i += blockDim.x * blockDim.y * blockDim.z) {
    dh_inv_[i] = dev_.dh_inv_[i];
  }

  __syncthreads();
  if (tid < ncoset(4))
    coefs_[tid] = coef_[tid];

  if (tid == 0) {
    // the cube center is initialy expressed in lattice coordinates but we
    // always do something like this. x = x + lower_corner + cube_center (+
    // roffset) - grid_lower_corner so shift the cube center already
    task.cube_center.z += task.lb_cube.z - dev_.grid_lower_corner_[0];
    task.cube_center.y += task.lb_cube.y - dev_.grid_lower_corner_[1];
    task.cube_center.x += task.lb_cube.x - dev_.grid_lower_corner_[2];

    if (distributed__) {
      if (task.apply_border_mask) {
        compute_window_size(
            dev_.grid_local_size_,
            dev_.tasks[dev_.first_task + blockIdx.x].border_mask,
            dev_.grid_border_width_, &task.window_size, &task.window_shift);
      }
    }
  }
  __syncthreads();

  for (int z = threadIdx.z; z < task.cube_size.z; z += blockDim.z) {
    int z2 = (z + task.cube_center.z) % dev_.grid_full_size_[0];

    if (z2 < 0)
      z2 += dev_.grid_full_size_[0];

    if (distributed__) {
      // check if the point is within the window
      if (task.apply_border_mask) {
        // this test is only relevant when the grid is split over several mpi
        // ranks. in that case we take only the points contributing to local
        // part of the grid.
        if ((z2 < task.window_shift.z) || (z2 > task.window_size.z)) {
          continue;
        }
      }
    }

    // compute the coordinates of the point in atomic coordinates
    T kremain;
    short int ymin = 0;
    short int ymax = task.cube_size.y - 1;

    if (orthorhombic_ && !task.apply_border_mask) {
      ymin = (2 * (z + task.lb_cube.z) - 1) / 2;
      ymin *= ymin;
      kremain = task.discrete_radius * task.discrete_radius -
                ((T)ymin) * dh_[8] * dh_[8];
      ymin = ceil(-1.0e-8 - sqrt(fmax(0.0, kremain)) * dh_inv_[4]);
      ymax = 1 - ymin - task.lb_cube.y;
      ymin = ymin - task.lb_cube.y;
    }

    for (int y = ymin + threadIdx.y; y <= ymax; y += blockDim.y) {
      int y2 = (y + task.cube_center.y) % dev_.grid_full_size_[1];
      if (y2 < 0)
        y2 += dev_.grid_full_size_[1];

      if (distributed__) {
        if (task.apply_border_mask) {
          // check if the point is within the window
          if ((y2 < task.window_shift.y) || (y2 > task.window_size.y)) {
            continue;
          }
        }
      }

      short int xmin = 0;
      short int xmax = task.cube_size.x - 1;
      if (orthorhombic_ && !task.apply_border_mask) {
        xmin = (2 * (y + task.lb_cube.y) - 1) / 2;
        xmin *= xmin;
        xmin =
            ceil(-1.0e-8 - sqrt(fmax(0.0, kremain - xmin * dh_[4] * dh_[4])) *
                               dh_inv_[0]);
        xmax = 1 - xmin - task.lb_cube.x;
        xmin = xmin - task.lb_cube.x;
      }

      for (int x = xmin + threadIdx.x; x <= xmax; x += blockDim.x) {
        int x2 = (x + task.cube_center.x) % dev_.grid_full_size_[2];

        if (x2 < 0)
          x2 += dev_.grid_full_size_[2];

        if (distributed__) {
          if (task.apply_border_mask) {
            // check if the point is within the window (only true or false
            // when using mpi) otherwise MPI=1 always true
            if ((x2 < task.window_shift.x) || (x2 > task.window_size.x)) {
              continue;
            }
          }
        }

        // I make no distinction between orthorhombic and non orthorhombic
        // cases

        T3 r3;
        if (orthorhombic_) {
          r3.x = (x + task.lb_cube.x + task.roffset.x) * dh_[0];
          r3.y = (y + task.lb_cube.y + task.roffset.y) * dh_[4];
          r3.z = (z + task.lb_cube.z + task.roffset.z) * dh_[8];
        } else {
          r3 = compute_coordinates(dh_, (x + task.lb_cube.x + task.roffset.x),
                                   (y + task.lb_cube.y + task.roffset.y),
                                   (z + task.lb_cube.z + task.roffset.z));
        }

        if (distributed__) {
          // check if the point is inside the sphere or not. Note that it does
          // not apply for the orthorhombic case when the full sphere is inside
          // the region of interest.

          if (((task.radius * task.radius) <=
               (r3.x * r3.x + r3.y * r3.y + r3.z * r3.z)) &&
              (!orthorhombic_ || task.apply_border_mask))
            continue;
        } else {
          // we do not need to do this test for the orthorhombic case
          if ((!orthorhombic_) && ((task.radius * task.radius) <=
                                   (r3.x * r3.x + r3.y * r3.y + r3.z * r3.z)))
            continue;
        }

        // allow computation of the address in parallel to starting the
        // computations
        // T *grid_elem =
        //     dev_.ptr_dev[1] +
        //     (z2 * dev_.grid_local_size_[1] + y2) * dev_.grid_local_size_[2] +
        //     x2;

        T res = coefs_[0];

        if (task.lp >= 1) {
          res += coefs_[1] * r3.x;
          res += coefs_[2] * r3.y;
          res += coefs_[3] * r3.z;
        }
        const T r3xy = r3.x * r3.y;
        const T r3xz = r3.x * r3.z;
        const T r3yz = r3.y * r3.z;
        const T r3x2 = r3.x * r3.x;
        const T r3y2 = r3.y * r3.y;
        const T r3z2 = r3.z * r3.z;

        if (task.lp >= 2) {
          res += coefs_[4] * r3x2;
          res += coefs_[5] * r3xy;
          res += coefs_[6] * r3xz;
          res += coefs_[7] * r3y2;
          res += coefs_[8] * r3yz;
          res += coefs_[9] * r3z2;
        }

        if (task.lp >= 3) {
          res += coefs_[10] * r3x2 * r3.x;
          res += coefs_[11] * r3x2 * r3.y;
          res += coefs_[12] * r3x2 * r3.z;
          res += coefs_[13] * r3.x * r3y2;
          res += coefs_[14] * r3xy * r3.z;
          res += coefs_[15] * r3.x * r3z2;
          res += coefs_[16] * r3y2 * r3.y;
          res += coefs_[17] * r3y2 * r3.z;
          res += coefs_[18] * r3.y * r3z2;
          res += coefs_[19] * r3z2 * r3.z;
        }

        if (task.lp >= 4) {
          res += coefs_[20] * r3x2 * r3x2;
          res += coefs_[21] * r3x2 * r3xy;
          res += coefs_[22] * r3x2 * r3xz;
          res += coefs_[23] * r3x2 * r3y2;
          res += coefs_[24] * r3x2 * r3yz;
          res += coefs_[25] * r3x2 * r3z2;
          res += coefs_[26] * r3xy * r3y2;
          res += coefs_[27] * r3xz * r3y2;
          res += coefs_[28] * r3xy * r3z2;
          res += coefs_[29] * r3xz * r3z2;
          res += coefs_[30] * r3y2 * r3y2;
          res += coefs_[31] * r3y2 * r3yz;
          res += coefs_[32] * r3y2 * r3z2;
          res += coefs_[33] * r3yz * r3z2;
          res += coefs_[34] * r3z2 * r3z2;
        }

        // beware it is coef_ (global memory) here not coefs_ (shared memory)
        if (task.lp > 4) {
          for (int ic = 35; ic < ncoset(task.lp); ic++) {
            auto &co = coset_inv[ic];
            T tmp = coef_[ic];
            for (int po = 0; po < co.l[2]; po++)
              tmp *= r3.z;
            for (int po = 0; po < co.l[1]; po++)
              tmp *= r3.y;
            for (int po = 0; po < co.l[0]; po++)
              tmp *= r3.x;
            res += tmp;
          }
        }

        atomicAdd(
            dev_.ptr_dev[1] +
                (z2 * dev_.grid_local_size_[1] + y2) *
                    dev_.grid_local_size_[2] +
                x2,
            res * exp(-(r3.x * r3.x + r3.y * r3.y + r3.z * r3.z) * task.zetp));
      }
    }
  }
  __syncthreads();
}
/*******************************************************************************
 * \brief Launches the Cuda kernel that collocates all tasks of one grid level.
 ******************************************************************************/
void context_info::collocate_one_grid_level(const int level,
                                            const enum grid_func func,
                                            int *lp_diff) {

  if (number_of_tasks_per_level_[level] == 0)
    return;

  // Compute max angular momentum.
  const ldiffs_value ldiffs = prepare_get_ldiffs(func);
  smem_parameters smem_params(ldiffs, lmax());

  *lp_diff = smem_params.lp_diff();
  init_constant_memory();

  // kernel parameters
  kernel_params params = set_kernel_parameters(level, smem_params);
  params.func = func;

  // Launch !
  const dim3 threads_per_block(4, 4, 4);

  if (func == GRID_FUNC_AB) {
    calculate_coefficients<double, true>
        <<<number_of_tasks_per_level_[level], threads_per_block,
           smem_params.smem_per_block(), level_streams[level]>>>(params);
  } else {
    calculate_coefficients<double, false>
        <<<number_of_tasks_per_level_[level], threads_per_block,
           smem_params.smem_per_block(), level_streams[level]>>>(params);
  }

  if (grid_[level].is_distributed()) {
    if (grid_[level].is_orthorhombic())
      collocate_kernel<double, double3, true, true>
          <<<number_of_tasks_per_level_[level], threads_per_block,
             ncoset(4) * sizeof(double), level_streams[level]>>>(params);
    else
      collocate_kernel<double, double3, true, false>
          <<<number_of_tasks_per_level_[level], threads_per_block,
             ncoset(4) * sizeof(double), level_streams[level]>>>(params);
  } else {
    if (grid_[level].is_orthorhombic())
      collocate_kernel<double, double3, false, true>
          <<<number_of_tasks_per_level_[level], threads_per_block,
             ncoset(4) * sizeof(double), level_streams[level]>>>(params);
    else
      collocate_kernel<double, double3, false, false>
          <<<number_of_tasks_per_level_[level], threads_per_block,
             ncoset(4) * sizeof(double), level_streams[level]>>>(params);
  }
}
} // namespace rocm_backend
#endif // defined(__OFFLOAD_HIP) && !defined(__NO_OFFLOAD_GRID)
